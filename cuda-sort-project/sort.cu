#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <assert.h>

#include "CycleTimer.h"

float toBW(int bytes, float sec) {
  return static_cast<float>(bytes) / (1024. * 1024. * 1024.) / sec;
}


__global__ void
enum_kernel(int N, int input[], int output[]) {



    //每个kernel用来确定一个input所在的位置。
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
    {
        int k=0;
        for (int j=0;j<N;j++)
        {
            if (input[index]>input[j])
            {
                k++;
            }
        }
        output[k]=input[index];
    }        
}

__device__ void merge(int l,int m,int r,int data[],int tmp[])
{
    int i=l,j=m,k=l;
    while (i<m&&j<r)
    {
        if (tmp[i]<=tmp[j])
        {
            data[k++]=tmp[i++];
        }
        else
        {
            data[k++]=tmp[j++];
        }
    }
    while (i<m) data[k++]=tmp[i++];
    while (j<r) data[k++]=tmp[j++];
}



__global__ void
merge_kernel(int N, int chunk,int data[],int tmp[]) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index>N) return ;
    //当数据量过多时，这里会发生溢出。暂时使用变负数。
    int start=index*chunk;
    if (start>=N || start<0) return ;

    int left=start;
    int mid=min(start+(int)(chunk/2),N);
    int right=min(start+chunk,N);
    // printf("l=%d,m=%d,r=%d\n",left,mid,right);
    // if (start<0) assert(0);
    merge(left, mid,right,data,tmp);
}

void enumSort(int N,int input[],int output[])
{
    hipSetDevice(1);
    int totalBytes = sizeof(int)* 2*N;
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    int *device_i;
    int *device_o;

    hipMalloc((void **)&device_i, N*sizeof(int));
    hipMalloc((void **)&device_o, N*sizeof(int));

    double startTime = CycleTimer::currentSeconds();

    hipMemcpy(device_i, input, N*sizeof(int), hipMemcpyHostToDevice);

    double startKernelTime = CycleTimer::currentSeconds();

    enum_kernel<<<blocks,threadsPerBlock>>>(N,device_i,device_o);    

    // hipDeviceSynchronize();
    hipDeviceSynchronize();
    double endKernelTime = CycleTimer::currentSeconds();
    hipMemcpy(output, device_o, N*sizeof(int), hipMemcpyDeviceToHost);
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double overallComputeDuration = endKernelTime - startKernelTime;
    printf("Comput time: %.3f ms\n", 1000.f * overallComputeDuration);
    printf("Overall time: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    hipFree(device_i);
    hipFree(device_o);
}


//cuda merge sort 就是chunk取2的幂次然后每个kernel会出里start+chunk的数据



void mergeSort(int N,int input[],int output[])
{
    hipSetDevice(1);
    int totalBytes = sizeof(int) * 3*N;
    // const int blocks = (((N + threadsPerBlock - 1) / threadsPerBlock))/2+1;

    int *device_i;
    int *tmp;
    hipMalloc((void **)&device_i, N*sizeof(int));
    hipMalloc((void **)&tmp, N*sizeof(int));

    double startTime = CycleTimer::currentSeconds();

    hipMemcpy(device_i, input, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tmp, input, N*sizeof(int), hipMemcpyHostToDevice);

    double startKernelTime = CycleTimer::currentSeconds();

    
    for (int chunk=2;chunk<2*N;chunk*=2)
    {
        // const int threadsPerBlock = 512;
        const int threadsPerBlock=1;
        const int blocks = ((N + threadsPerBlock*chunk - 1) / (threadsPerBlock*chunk));
        merge_kernel<<<blocks,threadsPerBlock>>>(N,chunk,device_i,tmp);
        hipDeviceSynchronize();
        merge_kernel<<<blocks,threadsPerBlock>>>(N,chunk,tmp,device_i);
        hipDeviceSynchronize();
    }
    

    double endKernelTime = CycleTimer::currentSeconds();
    hipMemcpy(output, device_i, N*sizeof(int), hipMemcpyDeviceToHost);
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double overallComputeDuration = endKernelTime - startKernelTime;
    printf("Comput time: %.3f ms\n", 1000.f * overallComputeDuration);
    printf("Overall time: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    hipFree(device_i);
    hipFree(tmp);
}






void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}